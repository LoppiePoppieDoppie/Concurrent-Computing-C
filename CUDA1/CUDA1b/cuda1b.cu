#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand.h"
#include <hiprand/hiprand_kernel.h>
#define DIM 1024

__global__ void func(float * x, float * A, float * result, int dim) {
	int i = threadIdx.x;
		for (int j = 0; j < dim; j++) {
			result[i] += A[i * dim + j] * x[j];
		}
}

void printMatrix(const char * name, float * matrix, int n, int m) {
	printf("%s = \n", name);
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < m; j++) {
			printf("%f ", matrix[i * m + j]);
		}
		printf("\n");
	}
}

void printVector(const char * name, float * array, int dim) {
	printf("%s = \n", name);
	for (int i = 0; i < dim; i++) {
		printf("%f ", array[i]);
	}
	printf("\n");
}

void zeros(float * array, int dim) {
	for (int i = 0; i < dim; i++) {
		array[i] = 0.;
	}
}

int main (int argc, const char ** args) {
	float *mul;
	float *d_x;
	float *d_A;
	float *d_result;
	float ms = 0;
	float sum = 0;
	
	hiprandGenerator_t gen;
	hipEvent_t start, end;
	
	mul = (float *) malloc(DIM * sizeof(float));
	hipMalloc((void **) &d_x, DIM * sizeof(float));	//Allocate DIM floats on device
	hipMalloc((void **) &d_A, DIM * DIM * sizeof(float));
	hipMalloc((void **) &d_result, DIM * sizeof(float));
	
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);	//Create pseudo-random number generator 
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);	//Set seed
	hiprandGenerateUniform(gen, d_x, DIM);	//Generate DIM floats
	hiprandGenerateUniform(gen, d_A, DIM * DIM);	//Generate DIM floats
	
	zeros(mul, DIM);	
	hipMemcpy(d_result, mul, DIM * sizeof(float), hipMemcpyHostToDevice);	//Copies data between Host and Device
		
	hipEventCreate(&start);	//Create an event object
	hipEventCreate(&end);	
	hipEventRecord(start);	//Record an event
	func<<<1, DIM>>>(d_x, d_A, d_result, DIM);
	hipEventRecord(end);
	
	hipMemcpy(mul, d_result, DIM * sizeof(float), hipMemcpyDeviceToHost);	//Copies data between Device and Host

	hipEventSynchronize(end);	//Wait until event complete
	hipEventElapsedTime(&ms, start, end);

	for(int i=0;i<DIM;i++)
	sum += mul[i];
		
	printf("Time is %f msec\n", ms);
	printf("Sum of elements = %f", sum);
	
	/* Cleanup */
	free(mul);
	
	hipFree(d_x);
	hipFree(d_A);
	hipFree(d_result);
	
	hiprandDestroyGenerator(gen);
	return 0;
}