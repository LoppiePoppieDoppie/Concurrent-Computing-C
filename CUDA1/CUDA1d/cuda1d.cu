#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand.h"
#include <hiprand/hiprand_kernel.h>
#define DIM 1024
#define tid threadIdx
#define bid blockIdx
#define bdim blockDim

__global__ void func(float * x, float * A, float * result) {
	__shared__ float shared_x[DIM];		//Use shared memory for x
	
	int i = bid.x;
	int j = tid.x;
	shared_x[j] = A[i * DIM +j]*shared_x[j];
	__syncthreads(); 	//Wait for all warps in a block to reach that point in your code
	
		for (unsigned int stride = bdim.x >> 1; stride > 0; stride >>= 1) {	
		__syncthreads();
		
		if (tid.x < stride) {
			shared_x[tid.x] += shared_x[tid.x + stride];
		}
	}
	
	if (tid.x == 0) {
		result[bid.x] = shared_x[0];
	}
	
}

void printMatrix(const char * name, float * matrix, int n, int m) {
	printf("%s = \n", name);
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < m; j++) {
			printf("%f ", matrix[i * m + j]);
		}
		printf("\n");
	}
}

void printVector(const char * name, float * array, int dim) {
	printf("%s = \n", name);
	for (int i = 0; i < dim; i++) {
		printf("%f ", array[i]);
	}
	printf("\n");
}

void zeros(float * array, int dim) {
	for (int i = 0; i < dim; i++) {
		array[i] = 0.;
	}
}

int main (int argc, const char ** args) {
	
	float *multiply;
	float *d_x;
	float *d_A;
	float *d_result;
	float ms = 0;
	float sum = 0.0;
	
	hiprandGenerator_t gen;
	hipEvent_t start, end;
	
	multiply = (float *) malloc(DIM * sizeof(float));
	hipMalloc((void **) &d_x, DIM * sizeof(float));	//Allocate DIM floats on device
	hipMalloc((void **) &d_A, DIM * DIM * sizeof(float));
	hipMalloc((void **) &d_result, DIM * sizeof(float));
	
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);	//Create pseudo-random number generator 
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);	//Set seed
	hiprandGenerateUniform(gen, d_x, DIM);	//Generate DIM floats
	hiprandGenerateUniform(gen, d_A, DIM * DIM);	//Generate DIM floats
	
	zeros(multiply, DIM);
	hipMemcpy(d_result, multiply, DIM * sizeof(float), hipMemcpyHostToDevice);		//Copies data between Host and Device

	hipEventCreate(&start);	//Create an event object
	hipEventCreate(&end);
	hipEventRecord(start);		//Record an event
	func<<<DIM, DIM>>>(d_x, d_A, d_result);
	hipEventRecord(end);
	
	hipMemcpy(multiply, d_result, DIM * sizeof(float), hipMemcpyDeviceToHost);		//Copies data between Device and Host

	hipEventSynchronize(end);	//Wait until event complete
	hipEventElapsedTime(&ms, start, end);

	for(int i=0;i<DIM;i++)
	sum += multiply[i];
	
	printf("Time is %f msec\n", ms);
	printf("Sum of elements = %f\n\n", sum);
	
	/* Cleanup */
	free(multiply);
	
	hipFree(d_x);
	hipFree(d_A);
	hipFree(d_result);
	
	hiprandDestroyGenerator(gen);
	return 0;
}