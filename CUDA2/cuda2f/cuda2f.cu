#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define block 1024
#define block_part 32
#define count 1024*1024
#define DIM count*block
#define tid threadIdx
#define bid blockIdx
#define bdim blockDim

#define CUDA_DEBUG

#ifdef CUDA_DEBUG

#define CUDA_CHECK_ERROR(err)           \
if (err != hipSuccess) {          \
printf("Cuda error: %s\n", hipGetErrorString(err));    \
printf("Error in file: %s, line: %i\n", __FILE__, __LINE__);  \
}                 \

#else

#define CUDA_CHECK_ERROR(err)

#endif

__inline__ __device__
float warpReduceSum(float res) {
  for (int offset = warpSize/2; offset > 0; offset /= 2) 
    res += __shfl_down_sync(0xffffffff,res, offset);	//exchange a variable between threads within a warp and get res from lane offset
  return res;
}

__inline__ __device__
float blockReduceSum(float res, float *sumv) {
	int lane = tid.x % warpSize;
	int wid = tid.x / warpSize;

	res = warpReduceSum(res);

	if (lane==0) sumv[wid]=res; 

	__syncthreads(); 
	if(tid.x < bdim.x / warpSize)
	{
		res = warpReduceSum(sumv[lane]);
	}

	return res;
}

__global__ void func(float * a, float * b, float * sum) {
	__shared__ float sumv[block_part];
	
	int i = bid.x * bdim.x + tid.x;
	float res = blockReduceSum(a[i]*b[i],sumv);

	if (tid.x == 0) 
	{
		sum[bid.x] = res;
	}
}

__global__ void vecSum(float *partsum,float *sum)
{	__shared__ float sumv[block_part];

	int i = bid.x * bdim.x + tid.x;
	float res = blockReduceSum(partsum[i],sumv);

	if (tid.x == 0) 
	{
		sum[bid.x] = res;
	}
}

void printVector(const char * name, float * array, int dim) {
	printf("%s = \n", name);
	for (int i = 0; i < dim; i++) {
		printf("%f ", array[i]);
	}
	printf("\n");
}

int main (int argc, const char ** args) {
	
	float *d_a;
	float *d_b;
	float *d_result;
	float *d_sum;
	float *d_partsum;
	float res = 0.0;
	float ms = 0.0;
	
	hiprandGenerator_t gen;
	hipEvent_t start, end;

	unsigned long size = sizeof(float) * DIM;
	size_t heapSize = size * 3;
	
	/* Set resource limits */
	CUDA_CHECK_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize));	//GPU malloc heap size
	
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);	//Create pseudo-random number generator 
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);	//Set seed
	
	CUDA_CHECK_ERROR(hipMalloc((void **) &d_a,size));	//Allocate DIM floats on device
	CUDA_CHECK_ERROR(hipMalloc((void **) &d_b,size));
	CUDA_CHECK_ERROR(hipMalloc((void **) &d_result,sizeof(float)));
	CUDA_CHECK_ERROR(hipMalloc((void **) &d_sum,sizeof(float)*count));
	CUDA_CHECK_ERROR(hipMalloc((void **) &d_partsum,sizeof(float)*count));
	
	hipMemcpy(d_result, &res, sizeof(float), hipMemcpyHostToDevice);		//Copies data between Host and Device
	hiprandGenerateUniform(gen, d_a, DIM);	//Generate DIM floats
	hiprandGenerateUniform(gen, d_b, DIM);

	hipEventCreate(&start);	//Create an event object
	hipEventCreate(&end);
	hipEventRecord(start);		//Record an event
	func<<<count, block>>>(d_a, d_b, d_partsum);
	vecSum<<<count / block, block>>>(d_partsum,d_sum);
	vecSum<<<1, block>>>(d_sum, d_result);
	hipEventRecord(end);
	
	hipEventSynchronize(end);		//Wait until event complete
	hipMemcpy(&res, d_result, sizeof(float), hipMemcpyDeviceToHost);		//Copies data between Device and Host
	hipEventElapsedTime(&ms, start, end);
	printf("Time is %f msec\n", ms);
	
	/* Cleanup */
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_result);
	hipFree(d_sum);
	hipFree(d_partsum);
	
	hiprandDestroyGenerator(gen);
	return 0;
}