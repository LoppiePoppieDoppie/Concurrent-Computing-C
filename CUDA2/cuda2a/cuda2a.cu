#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define block 1024
#define count 1024*1024
#define DIM count*block

void func(float * a, float * b, double * result) {
	for(long i = 0; i < DIM; i++)
		*result += a[i] * b[i];
}

/*
void printVector(const char * name, float * array, int dim) {
	printf("%s = \n", name);
	for (int i = 0; i < dim; i++) {
		printf("%f ", array[i]);
	}
	printf("\n");
}
*/

int main (int argc, const char ** args) {
	
	float *a;
	float *b;
	double result = 0.0;
	float ms = 0.0;
	
	hiprandGenerator_t gen;
	hipEvent_t start, end;
	
	a = (float*)calloc(sizeof(float), DIM);
	b = (float*)calloc(sizeof(float), DIM);
	
	hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);	//Create pseudo-random number generator
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);	//Set seed
	hiprandGenerateUniform(gen, a, DIM);		//Generate DIM floats
	hiprandGenerateUniform(gen, b, DIM);
	
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);	//Create an event object
	func(a, b, &result);
	hipEventRecord(end);	//Record an event
	
	hipEventSynchronize(end);		//Wait until event complete
	hipEventElapsedTime(&ms, start, end);

	printf("The result of multiplication is %f\n" , result);
	printf("Time is %f msec\n", ms);
	
	/* Cleanup */
	free(a);
	free(b);
	
	hiprandDestroyGenerator(gen);
	return 0;
}