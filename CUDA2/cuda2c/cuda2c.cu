#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define block 1024
#define count 1024*1024
#define DIM count*block
#define tid threadIdx
#define bid blockIdx
#define bdim blockDim

#define CUDA_DEBUG

#ifdef CUDA_DEBUG

#define CUDA_CHECK_ERROR(err)           \
if (err != hipSuccess) {          \
printf("Cuda error: %s\n", hipGetErrorString(err));    \
printf("Error in file: %s, line: %i\n", __FILE__, __LINE__);  \
}                 \

#else

#define CUDA_CHECK_ERROR(err)

#endif

__global__ void func(float * a, float * b, float * result) {
	__shared__ float mulv[block];	//Use shared memory for mulv
	
	int i = bid.x * bdim.x + tid.x;
	mulv[tid.x] = a[i] * b[i];
	__syncthreads();	//Wait for all warps in a block to reach that point in your code
	
	if(tid.x == 0){
		float res = 0;
		
		for(int i = 0;i < block; i++){
			res += mulv[i];
		}
		atomicAdd(result, res);
	}
}

void printVector(const char * name, float * array, int dim) {
	printf("%s = \n", name);
	for (int i = 0; i < dim; i++) {
		printf("%f ", array[i]);
	}
	printf("\n");
}
int main (int argc, const char ** args) {
	
	float *d_a;
	float *d_b;
	float *d_result;
	float res = 0.0;
	float ms = 0.0;
	
	hiprandGenerator_t gen;
	hipEvent_t start, end;

	unsigned long size = sizeof(float) * DIM;
	size_t heapSize = size * 3;
	
	/* Set resource limits */
	CUDA_CHECK_ERROR(hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize));	//GPU malloc heap size
	
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);	//Create pseudo-random number generator 
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);	//Set seed
	
	CUDA_CHECK_ERROR(hipMalloc((void **) &d_a,size));		//Allocate DIM floats on device
	CUDA_CHECK_ERROR(hipMalloc((void **) &d_b,size));
	CUDA_CHECK_ERROR(hipMalloc((void **) &d_result,sizeof(float)));
	
	hipMemcpy(d_result, &res, sizeof(float), hipMemcpyHostToDevice);		//Copies data between Host and Device
	
	hiprandGenerateUniform(gen, d_a, DIM);	//Generate DIM floats
	hiprandGenerateUniform(gen, d_b, DIM);
	
	hipEventCreate(&start);	//Create an event object
	hipEventCreate(&end);
	hipEventRecord(start);		//Record an event
	func<<<count,block>>>(d_a, d_b, d_result);
	hipEventRecord(end);
	
	hipEventSynchronize(end);		//Wait until event complete
	hipMemcpy(&res, d_result, sizeof(float), hipMemcpyDeviceToHost);		//Copies data between Device and Host
	hipEventElapsedTime(&ms, start, end);
	
	printf("Time is %f msec\n", ms);
	
	/* Cleanup */
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_result);
	
	hiprandDestroyGenerator(gen);
	return 0;
}