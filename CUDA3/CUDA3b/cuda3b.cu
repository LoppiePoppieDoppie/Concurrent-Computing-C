#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define DIM 4096
#define block 32

__global__ void func(float * A, float * A_tr) {
	int x = blockIdx.x * block + threadIdx.x;
	int y = blockIdx.y * block + threadIdx.y;
	
	int i = gridDim.x * block;
	int j = gridDim.y * block;
	
	A_tr[x * j + y] = A[y * i + x];
}

int main (int argc, const char ** args) {
	
	float *d_A;
	float *d_A_tr;
	float ms = 0.0;
	
	hiprandGenerator_t gen;
	hipEvent_t start, end;

	hipMalloc((void **) &d_A, DIM * DIM * sizeof(float));		//Allocate DIM floats on device
	hipMalloc((void **) &d_A_tr, DIM * DIM * sizeof(float));
	
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);	//Create pseudo-random number generator 
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);	//Set seed
	hiprandGenerateUniform(gen, d_A, DIM * DIM);		//Generate DIM floats
	
	hipEventCreate(&start);
	hipEventCreate(&end);
	
	/*kernel config*/
	dim3 threads(block, block);		//block size
    dim3 grid(DIM / threads.x, DIM / threads.y);
	
	hipEventRecord(start);		//Create an event object
	func<<<grid,threads>>>(d_A, d_A_tr);
	hipEventRecord(end);	//Record an event
	
	hipEventSynchronize(end);	//Wait until event complete
	hipEventElapsedTime(&ms, start, end);
	printf("Time is %f msec\n", ms);
	//printf("Sum of elements = %f\n\n", sum);
	/* Cleanup */
	hipFree(d_A);
	hipFree(d_A_tr);
	
	hiprandDestroyGenerator(gen);
	return 0;
}
