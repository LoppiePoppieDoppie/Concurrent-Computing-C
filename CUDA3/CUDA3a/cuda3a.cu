#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define DIM 4096

void func(float * A, float * Atr) {
	for (int i = 0; i < DIM; i++) {
		for (int j = 0; j < DIM; j++) {
			Atr[j * DIM + i] = A[i * DIM + j];
		}
	}
}

void printMatrix(const char * name, float * matrix, int n, int m) {
	printf("%s = \n", name);
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < m; j++) {
			printf("%f ", matrix[i * m + j]);
		}
		printf("\n");
	}
}

int main (int argc, const char ** args) {
	
	float *A;
	float *Atr;
	float ms = 0.0;

	hiprandGenerator_t gen;
	hipEvent_t start, end;
	
	A = (float *) malloc(DIM * DIM * sizeof(float));	//Allocate DIM floats on device
	Atr = (float *) malloc(DIM * DIM * sizeof(float));
	
	hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);	//Create pseudo-random number generator
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);	//Set seed
	hiprandGenerateUniform(gen, A, DIM * DIM);	//Generate DIM floats
	
	hipEventCreate(&start);	//Create an event object
	hipEventCreate(&end);
	hipEventRecord(start);		//Record an event
	func(A, Atr);
	hipEventRecord(end);
	
	hipEventSynchronize(end);		//Wait until event complete
	hipEventElapsedTime(&ms, start, end);
	
	printf("Time is %f msec\n", ms);
	//printf("Sum of elements = %f\n\n", sum);
	/* Cleanup */
	free(A);
	free(Atr);
	
	hiprandDestroyGenerator(gen);
	return 0;
}